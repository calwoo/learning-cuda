#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>

__global__ void rgb_to_grayscale_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    const int channels = 3;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int inputOffset = (row * width + col) * channels;
        int outputOffset = row * width + col;

        unsigned char r = input[inputOffset];
        unsigned char g = input[inputOffset + 1];
        unsigned char b = input[inputOffset + 2];

        output[outputOffset] = (unsigned char)(0.21 * r + 0.71 * g + 0.07 * b);
    }
}

// helper function for ceiling unsigned integer division
inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}

torch::Tensor rgb_to_grayscale(torch::Tensor image) {
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);

    const auto height = image.size(0);
    const auto width = image.size(1);

    auto result = torch::empty({height, width, 1}, torch::TensorOptions().dtype(torch::kByte).device(image.device()));

    dim3 threads_per_block(16, 16);     // using 256 threads per block
    dim3 number_of_blocks(cdiv(width, threads_per_block.x),
                          cdiv(height, threads_per_block.y));

    rgb_to_grayscale_kernel<<<number_of_blocks, threads_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<unsigned char>(),
        result.data_ptr<unsigned char>(),
        width,
        height
    );

    // check CUDA error status (calls hipGetLastError())
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
}
