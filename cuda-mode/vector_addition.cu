#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float *A_h, float *B_h, float *C_h, int n) {
    // initialize device memory
    float *A_d, *B_d, *C_d;
    size_t size = n * sizeof(float);

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // run kernel
    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

    // move output from device
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    const int n = 1000;
    float A_h[n];
    float B_h[n];
    float C_h[n];

    // generate some dummy vectors to add
    for (int i = 0; i < n; i += 1) {
        A_h[i] = float(i);
        B_h[i] = A_h[i] / 1000.0f;
    }

    vecAdd(A_h, B_h, C_h, n);

    for (int i = 0; i < n; i += 1) {
        if (i > 0) {
            printf(", ");
            if (i % 10 == 0) {
                printf("\n");
            }
        }
        printf("%8.3f", C_h[i]);
    }
    printf("\n");
}