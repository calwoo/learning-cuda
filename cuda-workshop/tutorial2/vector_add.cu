#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>



#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // allocate host memory
    a = (float*) malloc(sizeof(float) * N);
    b = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    // initialize host data
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // transfer data to device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // execute kernel
    vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
    return 0;
}
