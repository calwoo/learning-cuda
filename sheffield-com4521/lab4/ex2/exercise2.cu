#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);



__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}


void vectorAddCPU(int *a, int *b, int *c, int max) {
	for (int i = 0; i < max; i++) {
		c[i] = a[i] + b[i];
	}
}

int validate(int *c, int *c_ref, int max) {
	int numErrors = 0;
	for (int i = 0; i < max; i++) {
		if (c[i] != c_ref[i]) {
			numErrors += 1;
			printf("error found\n");
		}
	}
	return numErrors;
}



int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
	vectorAdd << <(N + THREADS_PER_BLOCK) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	vectorAddCPU(a, b, c_ref, N);

	errors = validate(c, c_ref, N);
	printf("num errors: %d\n", errors);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
