#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 512
#define M 256
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);



__global__ void matrixAdd(int *a, int *b, int *c, int numRows, int numCols) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < numRows) && (j < numCols)) {
        c[i*numCols+j] = a[i*numCols+j] + b[i*numCols+j];
    }
}


void matrixAddCPU(int *a, int *b, int *c, int numRows, int numCols) {
	for (int i = 0; i < numRows * numCols; i++) {
		c[i] = a[i] + b[i];
	}
}

int validate(int *c, int *c_ref, int numRows, int numCols) {
	int numErrors = 0;
	for (int i = 0; i < numRows * numCols; i++) {
		if (c[i] != c_ref[i]) {
			numErrors += 1;
			printf("error found: %d != %d\n", c[i], c_ref[i]);
		}
	}
	return numErrors;
}



int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * M * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
    dim3 blocksPerGrid(N/16, M/16, 1);
    dim3 threadsPerBlock(16, 16, 1);
	matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, M);
	checkCUDAError("CUDA kernel");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	matrixAddCPU(a, b, c_ref, N, M);

	errors = validate(c, c_ref, N, M);
	printf("num errors: %d\n", errors);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N * M; i++){
		a[i] = rand();
	}
}
